#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <unistd.h>
#include <stdint.h>
#include <inttypes.h>
#include "microbench_cuda.h"
#define BLOCK_SIZE 1024


extern "C" 
uint64_t *baseline_double_cuda(uint64_t * _array, unsigned long int size)
{
     double *d_array, *d_output;
     hipMalloc((void **) &d_array, sizeof(double)*size);
     hipMalloc((void **) &d_output, sizeof(double)*size);
     hipMemcpy(d_array, _array, sizeof(double)*size, hipMemcpyHostToDevice);

     unsigned int grid_cols = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;

     // Launch kernel 
     baseline_double_cuda_init<<<grid_cols, BLOCK_SIZE>>>(d_array, size);
     for(int j=0;j<3;j++)
     {
         baseline_double_cuda_kernel<<<grid_cols, BLOCK_SIZE>>>(d_array, d_output, size, j);
         hipDeviceSynchronize();
         hipMemcpy(d_array, d_output, sizeof(double)*size, hipMemcpyDeviceToDevice);
     }
     // Transefr results from device to host 
     hipMemcpy(_array, d_output, sizeof(double)*size, hipMemcpyDeviceToHost);
     hipFree(d_output);
     hipFree(d_array);
     return _array;
}
__global__ void baseline_double_cuda_kernel(double *input, double *output, int size, int j) 
{
    unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;

    if (pos < size)
    {
        output[pos] = input[pos]+pos*(j+1);
    }
}
__global__ void baseline_double_cuda_init(double *array, int size) 
{
    unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;

    if (pos < size) 
    {
        array[pos] = 0;
    }
}
